
#include <hip/hip_runtime.h>


#ifdef __NVCC__

//K in parallel
template < class U >
__global__ void extractMin(unsigned int* PQ, unsigned int* PQ_size, int* expandNodes,int* expandNodes_size,U* Cx,int* openList,int N,int K){
    
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    
    if(id<K && PQ_size[id]>0){

        //extract min from PQ
        int front = id* ( (N+K-1)/K );
        int node = PQ[front];

        // restructure the heap
        PQ[front]=PQ[front+PQ_size[id]-1];
        PQ_size[id]-=1;
        int pqIndex = 0;

        while(2*pqIndex+1 < PQ_size[id]){
            if(2*pqIndex+2 >= PQ_size[id]){
                if( Cx[PQ[front+pqIndex]] > Cx[PQ[front+2*pqIndex+1]]){
                    int swap = PQ[front + 2*pqIndex+1];
                    PQ[front + 2*pqIndex+1] = PQ[front +pqIndex];
                    PQ[front + pqIndex] = swap;
                    pqIndex = 2*pqIndex+1;
                }
                else
                    break;
            }
            else{
                if( Cx[PQ[front+pqIndex]] > Cx[PQ[front+2*pqIndex+1]] && Cx[PQ[front+2*pqIndex+1]] <= Cx[PQ[front+2*pqIndex+2]] ){
                    int swap = PQ[front + 2*pqIndex+1];
                    PQ[front + 2*pqIndex+1] = PQ[front +pqIndex];
                    PQ[front + pqIndex] = swap;
                    pqIndex = 2*pqIndex+1;
                }
                else if(Cx[PQ[front+pqIndex]] > Cx[PQ[front+2*pqIndex+2]] && Cx[PQ[front+2*pqIndex+2]] <= Cx[PQ[front+2*pqIndex+1]] ){
                    int swap = PQ[front + 2*pqIndex+2];
                    PQ[front + 2*pqIndex+2] = PQ[front +pqIndex];
                    PQ[front + pqIndex] = swap;
                    pqIndex = 2*pqIndex+2;                    
                } 
                else{
                    break;
                }
            }
            
        }

        //removed from openList
        openList[node] = -1;

        //added to expand next
        int len = atomicAdd(expandNodes_size,1);
        expandNodes[len]=node;
    }

} 


//for K in parallel

template < class T, class U >
__global__ void A_star_expand(int* off,int* edge,T* W, U* Hx,int* parent,volatile U* Cx,
    int* expandNodes,int* expandNodes_size, int* lock ,int* flagfound,int* openList,
    int N,int E, int K,int dest,int* nVFlag ){
       
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    
    if(id< *expandNodes_size ){

        int node = expandNodes[id];
        
        //reach dest
        if(node == dest){
            atomicOr(flagfound,1);
        }

        // expand
        int start = off[node];
        int end = E;
        if(node!=N-1)
            end = off[node+1];
        
        while(start < end){ 
            int child = edge[start];
            
            //deleted edges
            if(child<0){
                start++;
                continue;
            }

            //array L initilaized with 0
            //get the lock for child to update C(x)
            //loop till acquire the lock
            bool leaveLoop = false;

            while(leaveLoop==false){

                if(atomicCAS(&lock[child],0,1)==0){
                    //critical section
                    if( Cx[child] > (Cx[node] - Hx[node])+ W[start]+ Hx[child] ){
                        Cx[child]  = (Cx[node] - Hx[node])+ W[start]+ Hx[child];
                        __threadfence();
                        parent[child] = node;
        
                        if(openList[child]==-1){
                            nVFlag[child]=1;
                            //add only once
                        }
                    }

                    //end critical section
                    leaveLoop = true;

                    atomicCAS(&lock[child],1,0);

                }

                __syncthreads();

            }

            start++;
        }
        
    }//end 

}


//K in parallel -- O(N)
template < class U >
__global__ void keepHeapPQ(unsigned int* PQ, unsigned int* PQ_size,U* Cx,int N,int K){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id < K && PQ_size[id] > 0){
        int front  = id*( (N+K-1)/K );
        int size = PQ_size[id];
        
        for(int i=front;i<front+size;i++){
            if(2*i+2 < front+size){
                int cost = Cx[PQ[i]];
                int costLeft = Cx[PQ[2*i+1]];
                int costRight = Cx[PQ[2*i+2]]; 
                if( cost > costLeft  ||  cost > costRight  ){
                    int index ;
                    if(costLeft <= costRight)
                        index = 2*i+1;
                    else
                        index = 2*i+2;
                    
                    while(index > front){
                        if( Cx[PQ[(index-1)/2]] > Cx[PQ[index]] ){
                            int swap = PQ[index];
                            PQ[index] = PQ[(index-1)/2];
                            PQ[(index-1)/2] = swap;
                            index = (index-1)/2;
                        }
                        else
                            break;
                    }
                }
            }
            else if(2*i+1 < front+size){
                if(Cx[PQ[i]] > Cx[PQ[2*i+1]]){
                    int index = 2*i+1;
                    while(index > front){
                        if( Cx[PQ[(index-1)/2]] > Cx[PQ[index]] ){
                            int swap = PQ[index];
                            PQ[index] = PQ[(index-1)/2];
                            PQ[(index-1)/2] = swap;
                            index = (index-1)/2;
                        }
                        else
                            break;
                    }
                }
            }
        }
    }
}

//N threads
__global__ void setNV(int* nextFlag,int* nextV,int* nvSize,int N){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id < N){
        if(nextFlag[id]==1){
            int index = atomicAdd(nvSize,1);
            nextV[index]=id;
        }
    }
}


//for K in parallel
template <class U >
__global__ void insertPQ(unsigned int* PQ,unsigned int* PQS,int* nextV,int* nVsize,U* Cx,int K,int N,int* openList){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id < K){

        int front = id*( (N+K-1)/K );
        int i = id;
        
        while(i<*nVsize){            
            //if not already present
            if(openList[nextV[i]]!=-1){
                i+=K;
                continue;
            }

            PQ[front+PQS[id]]= nextV[i];
            PQS[id]+=1;

            //add in openList
            openList[nextV[i]] = id;

            if(PQS[id]>1){
                int index = PQS[id]-1;
                while(index>0){
                    if(Cx[PQ[front+ (index-1)/2]] > Cx[PQ[front+index]]){
                        int swap = PQ[front+index];
                        PQ[front+index]=PQ[front+ (index-1)/2];
                        PQ[front+ (index-1)/2] = swap;
                        index = (index-1)/2;
                    }
                    else
                        break;
                }
            }
            i += K;
        }
    }
}


//for K in parallel

template < class U >
__global__ void checkMIN(unsigned int* PQ, unsigned int* PQ_size,int* flagEnd,U* Cx,int dest,int N,int K){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    
    if(id < K && PQ_size[id] > 0 ){
        int front = id* ( (N+K-1)/K );
        int node = PQ[front];
        //check if atleast one min, dont end the a*
        if( Cx[node] < Cx[dest] ){
            atomicAnd(flagEnd,0);
        }
    }
}

template <class U>
__global__ void getCx(U* Cx,int dest,U* val){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id==0){
        *val = Cx[dest];
    }
}



#endif
