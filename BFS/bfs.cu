#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <limits.h>
#include <stdbool.h>

#define  MAX_EDGE  100000000
 

__global__ void BFS(int* off,int* edge,int* current,int* size,int N,int E,int* c_arr,int* c_size,int* dist){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id < *size){
        // printf("inside kernel %d %d\n",id,*size);
        int node = current[id];
        //extend this node
        int start = off[node];
        int end = E;
        if(node!=N-1)
            end = off[node+1];
        
        while(start<end){
            // add in list atomically in c_arr
            int child = edge[start];
            // printf("c %d \n",child);
            if ( dist[child] < 0){
                dist[child] = dist[node] + 1;
                int index = atomicAdd(c_size,1);
                c_arr[index]= child;
            }
            start++;  
        }

        // printf("s %d\n",*c_size);
    }
}


int main(){
    int N;
    scanf("%d\n",&N);
    int startNode;
    scanf("%d\n",&startNode);

    int* H_offset = (int*)malloc(sizeof(int)*N);
    int* H_edges  = (int*)malloc(sizeof(int)*MAX_EDGE);
    

    memset(H_offset,-1,sizeof(int)*N);

    int a,b;
    int prev_node = -1;
    int edge_size = 0;
    while(scanf("%d %d\n",&a,&b)!=EOF){
        if(a==prev_node){
            H_edges[edge_size]=b;
            edge_size++;
        }
        else{
            H_offset[a]=edge_size;
            H_edges[edge_size]=b;
            edge_size++;
            prev_node = a;

        }
    }

    for(int i=0;i<N;i++){
        if(H_offset[i]==-1){
            int j = i+1;
            int flag = 0;
            while(j<N){
                if(H_offset[j]==-1){
                }
                else{
                    H_offset[i]=H_offset[j];
                    flag= 1;
                    break;
                }
                j++;
            }
            if(flag==0){
                H_offset[i] = edge_size;
            }
        }
    }


    printf("completed input\n");

    int* H_current_node = (int*)malloc(sizeof(int)*edge_size);
    H_current_node[0]=startNode;
    int* H_c_size = (int*)malloc(sizeof(int));
    *H_c_size = 1;
    int* H_visited = (int*)malloc(sizeof(int)*N);
    memset(H_visited,-1,sizeof(int)*N);
    H_visited[startNode]=0;

    int* a0 = (int*)malloc(sizeof(int));
    *a0=0;

    int* a1 = (int*)malloc(sizeof(int));
    *a1=1;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int* D_offset;
    int* D_edges;
    int* D_visited;

    int* D_current_node1;
    int* D_c_size1;
    int* D_current_node2;
    int* D_c_size2;

    hipMalloc(&D_offset,sizeof(int)*N);
    hipMalloc(&D_visited,sizeof(int)*N);
    hipMalloc(&D_edges,sizeof(int)*edge_size);
    hipMalloc(&D_current_node1,sizeof(int)*edge_size);
    hipMalloc(&D_c_size1,sizeof(int));
    hipMalloc(&D_current_node2,sizeof(int)*edge_size);
    hipMalloc(&D_c_size2,sizeof(int));


    hipMemcpy(D_offset,H_offset,sizeof(int)*N,hipMemcpyHostToDevice);
    hipMemcpy(D_edges,H_edges,sizeof(int)*edge_size,hipMemcpyHostToDevice);
    hipMemcpy(D_current_node1,H_current_node,sizeof(int)*edge_size,hipMemcpyHostToDevice);
    hipMemcpy(D_visited,H_visited,sizeof(int)*N,hipMemcpyHostToDevice);
    hipMemcpy(D_c_size1,a1,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(D_c_size2,a0,sizeof(int),hipMemcpyHostToDevice);



    int i=1;
    hipEventRecord(start);
    while(*H_c_size>0){
        int numThreads = 512;
        int numBlocks = (*H_c_size+numThreads-1)/numThreads;


        if(i%2==1){
            //use array 1
            BFS<<<numThreads,numBlocks>>>(D_offset,D_edges,D_current_node1,D_c_size1,N,edge_size,D_current_node2,D_c_size2,D_visited);

            hipMemcpy(H_c_size,D_c_size2, sizeof(int),hipMemcpyDeviceToHost);
            // reset the index
            hipMemcpy(D_c_size1,a0,sizeof(int),hipMemcpyHostToDevice);
          
        }
        else{
            //use array 2
            BFS<<<numThreads,numBlocks>>>(D_offset,D_edges,D_current_node2,D_c_size2,N,edge_size,D_current_node1,D_c_size1,D_visited);
            
            hipMemcpy(H_c_size,D_c_size1, sizeof(int),hipMemcpyDeviceToHost);
            //reset index
            hipMemcpy(D_c_size2,a0,sizeof(int),hipMemcpyHostToDevice);

        }
        i++;
        
    }
    hipEventRecord(stop);
    
    hipMemcpy(H_visited,D_visited, sizeof(int)*N,hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    int max_level = 0;
    for(int i=0;i<N;i++){
        if(H_visited[i]>max_level){
            max_level = H_visited[i];
        }
        // printf("%d, %d\n",i,H_visited[i]);
    }
    
    printf("max-level: %d\n",max_level);
    printf("time: %f\n",milliseconds);

    return 0;
}
